#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstring>
#include <iomanip>
#include <numeric>
#include <stdlib.h>
#include <math.h>

using namespace std;

#define USETNS1


typedef float T;

typedef struct {
	unsigned int blocks;
	unsigned int blockSize;
	unsigned int Nx;
	unsigned int Ny;
	unsigned int size;
	unsigned int steps;
	unsigned int maxiter;
	T dt;
	T maxres;
	unsigned int nRowsZ;
} Parameters;


typedef struct {
	T rho;
	T cp;
	T k;
} MaterialProperties;

Parameters params;

MaterialProperties steel;
MaterialProperties Ag;
MaterialProperties MgO;
MaterialProperties inconel;
MaterialProperties NiCr;

#include "cpuFunctions.h"
#include "cudaFunctions.h"

int main(void)
{
    // Parameters
	params.blocks = 128;
	params.blockSize = 128;  
	params.Nx  = 1024;
	params.Ny  = 1024;
	params.size = params.Nx*params.Ny + 2*params.Nx;
	params.steps  = 1;
	params.maxiter  = 1000000;
	params.dt = 0.003333333333;    // 0.0033333333333333;
	params.maxres = 1e-3;
	params.nRowsZ = 32;    // number of rows for one deflation vector
	
	// steel
	steel.rho = 7610.0; // 7700
    steel.cp  = 545.0; // 560
    steel.k   = 21.0;

    // Ag
    Ag.rho = 8957.0;
    Ag.cp  = 362.0; // 368
    Ag.k   = 120; // 111.5

    // MgO
    MgO.rho = 3150.0;
    MgO.cp  = 1110.0; // 1140
    MgO.k   = 11.5;   // 10
    
    // inconel
    inconel.rho = 8470.0;
    inconel.cp  = 500.0; // 520
    inconel.k   = 20.5; 
    
    // NiCr
    NiCr.rho = 8200.0;
    NiCr.cp  = 528.0;
    NiCr.k   = 24.5;
    
    T t = 0; // time
    T totalIter = 0;
    
    
    dim3 dimGrid(params.blocks);
    dim3 dimBlock(params.blockSize);
	
	cout << "example 1: inverse task (Pohanka)" << endl;
	
	cpuInit(params.blocks, params.Nx, params.Ny);
	//cpuInitDeflation(params.nRowsZ, params.Nx, params.Ny);
	
	
	readGeometry(hm, params.Nx, params.Ny);	// materials
	
	readCoords(xc, dx, "xCoords1024.txt");
	readCoords(yc, dy, "yCoords1024.txt");
	
	readBC(tHF, params.steps);
	
	initX(hT, params.Nx, params.Ny);
	initA(params.dt, params.Nx, params.Ny);
	
	//initAZ(params.dt, params.nRowsZ, params.Nx, params.Ny);
	//initE(params.nRowsZ, params.Nx, params.Ny);
	
	//saveData<int>(hm, "materials1024", params.Nx, params.Ny);
	//saveDataInTime(hT, t, "temperature1024_res1e-3_TNS1", params.Nx, params.Ny);
	
	//check_var(dy, params.Nx, params.Ny);

	// CUDA
	cudaInit(hT, hV, hcc, hss, hww, hqB, params.blocks, params.Nx, params.Ny);
	
#ifdef USETNS1
	//makeTNS1<<<1024,1024>>>(kcc, kss, kww, dcc, dss, dww, params.Nx, params.Ny);
	makeTNS1f<<<1024,1024>>>(kcc, kss, kww, k3, dcc, dss, dww, params.Nx, params.Ny);
#endif	
	
	hipEvent_t startT, stopT;
	float elapsedTime;
	hipEventCreate(&startT);
	hipEventCreate(&stopT);
	hipEventRecord(startT,0);

	for (int miter=0; miter<params.steps; miter++) {

		hipMemcpy(dr, dT, sizeof(T)*params.size, hipMemcpyDeviceToDevice);	// r = rhs
		elementWiseMul<<<1024,1024>>>(dr, dV, params.Nx);	// r = V*r
		
		// add Neumann boundary here ... r = r + NeumannBC (dqB)
		//addNeumannBC<<<1,1024>>>(dr, dqB, (T)-1.0e6, params.Nx);   // constant
		addNeumannBC<<<1,1024>>>(dr, dqB, tHF[miter], params.Nx);   // time dependent


		SpMVv1<<<1024,1024>>>(dq, dcc, dss, dww, dT, params.Nx);   // q = Ax (version 1)

		AXPY<<<1024,1024>>>(dr, dq, (T)-1., (T)1., params.Nx);   // r = r - q

#ifdef USETNS1
		//SpMVv1<<<1024,1024>>>(dz, kcc, kss, kww, dr, params.Nx);  // z = M^(-1)r (version 1)
		SpMVv2<<<1024,1024>>>(dz, kcc, kss, k3, kww, dr, params.Nx);  // z = M^(-1)r (version 2)
		DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(drh, dr, dz, params.Nx, params.Ny);
#else
		DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(drh, dr, dr, params.Nx, params.Ny);
#endif
		hipMemcpy(&rhNew, drh, 1*sizeof(T), hipMemcpyDeviceToHost);
		hipMemset(drh,0,sizeof(T)); // reset;
		stop = rhNew * params.maxres * params.maxres;
		iter = 0;

		//cout << "stop:" << stop << ", residual: " << rhNew << endl;
		
		while (rhNew > stop && iter < params.maxiter) {

			iter++;
			totalIter++;
			//cout << "iteration:" << iter << ", residual: " << rhNew << endl;

			if (iter==1) {
#ifdef USETNS1
				hipMemcpy(dp, dz, sizeof(T)*params.size,hipMemcpyDeviceToDevice);
#else
				hipMemcpy(dp, dr, sizeof(T)*params.size,hipMemcpyDeviceToDevice);
#endif
			}
			else {
				bt = rhNew/rhOld;
#ifdef USETNS1	
				AXPY<<<1024,1024>>>(dp, dz, (T)1., bt, params.Nx);   // p = z + beta*p	
#else
				AXPY<<<1024,1024>>>(dp, dr, (T)1., bt, params.Nx);   // p = r + beta*p	
#endif

			}


			SpMVv1<<<1024,1024>>>(dq, dcc, dss, dww, dp, params.Nx);  // q = Ap (version 1)
			DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(dsg, dp, dq, params.Nx, params.Ny);   // sigma = <p,q>
			hipMemcpy(&sg, dsg, 1*sizeof(T), hipMemcpyDeviceToHost);
			hipMemset(dsg,0,sizeof(T)); // reset;
			ap = rhNew/sg;	// alpha = rhoNew / sigma
			AXPY<<<1024,1024>>>(dr, dq, -ap, (T)1., params.Nx);   // r = r - alpha*q
			AXPY<<<1024,1024>>>(dT, dp,  ap, (T)1., params.Nx);   // x = x + alpha*p

#ifdef USETNS1
			//SpMVv1<<<1024,1024>>>(dz, kcc, kss, kww, dr, params.Nx);  // z = M^(-1)r (version 1)
			SpMVv2<<<1024,1024>>>(dz, kcc, kss, k3, kww, dr, params.Nx);  // z = M^(-1)r (version 2)
#endif

			rhOld = rhNew;

#ifdef USETNS1
			DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(drh, dr, dz, params.Nx, params.Ny);   // rhoNew = <r,z>		
#else
			DOTGPU<T,128><<<dimGrid,dimBlock,params.blockSize*sizeof(T)>>>(drh, dr, dr, params.Nx, params.Ny);   // rhoNew = <r,r>
#endif
			hipMemcpy(&rhNew, drh, 1*sizeof(T), hipMemcpyDeviceToHost);
			hipMemset(drh,0,sizeof(T)); // reset;	
			

		}
		t += params.dt;
		cout << "time: " << t << " ,timestep:" << miter << " ,iteration:" << iter << endl;
		
		//if ((miter+1)%4000==0)
		//{
		//hipMemcpy(hT, dT, sizeof(T)*params.size, hipMemcpyDeviceToHost);
		//saveDataInTime(hT, t, "temperature1024_res1e-3_TNS1float", params.Nx, params.Ny);
		//}

	}
	
	hipEventRecord(stopT,0);
	hipEventSynchronize(stopT);
	hipEventElapsedTime(&elapsedTime, startT, stopT);
	cout<< "ellapsed time (cuda): " << elapsedTime << " miliseconds"	<< endl;
	
	cout << "Simulation finished." << endl;
	cout << "total number of iterations: " << totalIter << endl;
	
	hipEventDestroy(startT);
	hipEventDestroy(stopT);
	cudaFinalize();
	cpuFinalize();
	cpuFinalizeDeflation();
	return 0;
}

